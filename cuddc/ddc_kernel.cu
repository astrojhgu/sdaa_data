#include <complex>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cstdio> // 使用 C++ 风格的头文件

using namespace std;
static constexpr float PI=3.14159265358979323846;

// DDC 处理所需的 GPU 资源
struct DDCResources
{
    int N;  // 每次追加的数据长度
    int M; // 累积多少块数据后计算
    int NDEC;// 下抽样倍率
    int K;   // 滤波器抽头数/下抽样倍率
    int16_t *d_indata; // 设备中的输入数据缓冲区
    hipFloatComplex *d_outdata; // 设备中的输出数据缓冲区
    hipFloatComplex *mixed_data; // 存放混频结果的缓冲区
    float *d_fir_coeffs; // 滤波器系数，位于gpu显存中
    int16_t *h_indata; // 输入数据缓冲区，位于RAM中
    int h_index; // 输入缓冲区尾端索引号，每次追加数据都放入此位置
};

// 复数乘法
__device__ static hipFloatComplex complex_mult(float a, float b, float c, float d)
{
    return make_hipFloatComplex(a * c - b * d, a * d + b * c);
}

__global__ void mix(int16_t *indata, hipFloatComplex *mixed_data, int offset, int N, int M, int lo_ch)
{
    int total_size=N*M;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < total_size)
    {
        float phase=-(float)(i%N)*(float)lo_ch/(float)N*2.0*PI;
        float lo_cos=cos(phase);
        float lo_sin=sin(phase);
        mixed_data[offset + i] = complex_mult(float(indata[i]), 0.0f, lo_cos, lo_sin);
    }
}

// 设备核函数：FIR 滤波并下抽样
__global__ void fir_filter(hipFloatComplex *mixed_data, hipFloatComplex *outdata, const float *fir_coeffs, int NDEC, int K, int total_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int output_index = i;
    int input_index = i * NDEC;

    if (output_index < total_size / NDEC)
    {
        hipFloatComplex sum = make_hipFloatComplex(0.0f, 0.0f);
        for (int j = 0; j < K * NDEC; j++)
        {
            sum = hipCaddf(sum, hipCmulf(make_hipFloatComplex(fir_coeffs[j], 0.0f), mixed_data[input_index + j]));
        }
        outdata[output_index] = sum;
    }
}

// 初始化 DDC 资源
extern "C" struct DDCResources* init_ddc_resources(int N, int M, int NDEC, int K, const float *fir_coeffs)
{
    DDCResources *res = (DDCResources *)malloc(sizeof(DDCResources));
    res->NDEC = NDEC;
    res->K = K;
    res->N=N;
    res->M=M;
    int buffer_size = M * N + NDEC * (K - 1);
    int fir_size = NDEC * K;

    hipError_t err = hipMalloc((void **)&res->d_indata, M * N * sizeof(int16_t));
    assert(err == hipSuccess);
    err = hipMalloc((void **)&res->d_outdata, (M * N / NDEC) * sizeof(hipFloatComplex));
    assert(err == hipSuccess);
    err = hipMalloc((void **)&res->mixed_data, buffer_size * sizeof(hipFloatComplex));
    assert(err == hipSuccess);
    err = hipMalloc((void **)&res->d_fir_coeffs, fir_size * sizeof(float));
    assert(err == hipSuccess);

    res->h_indata = (int16_t *)malloc(M * N * sizeof(int16_t));
    assert(res->h_indata);
    res->h_index = 0;

    err = hipMemcpy(res->d_fir_coeffs, fir_coeffs, fir_size * sizeof(float), hipMemcpyHostToDevice);
    assert(err == hipSuccess);
    return res;
}

// 释放资源
extern "C" void free_ddc_resources(DDCResources *res)
{
    hipFree(res->d_indata);
    hipFree(res->d_outdata);
    hipFree(res->mixed_data);
    hipFree(res->d_fir_coeffs);
    free(res->h_indata);
    free(res);
}

// DDC 处理
extern "C" int ddc(const int16_t *indata, int lo_ch, DDCResources *res)
{
    memcpy(res->h_indata + res->h_index, indata, res->N * sizeof(int16_t));
    res->h_index += res->N;

    if (res->h_index == res->M * res->N)
    {
        int total_size = res->M * res->N;
        //int buffer_size = total_size + res->NDEC * (res->K - 1);
        int offset = res->NDEC * (res->K - 1);

        hipMemcpy(res->d_indata, res->h_indata, total_size * sizeof(int16_t), hipMemcpyHostToDevice);
        mix<<<(total_size + 255) / 256, 256>>>(res->d_indata, res->mixed_data, offset, res->N, res->M, lo_ch);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            return -1;
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess)
            return -1;

        fir_filter<<<(total_size / res->NDEC + 255) / 256, 256>>>(res->mixed_data, res->d_outdata, res->d_fir_coeffs, res->NDEC, res->K, total_size);
        err = hipGetLastError();
        if (err != hipSuccess)
            return -1;
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess)
            return -1;

        
        res->h_index = 0;
        return 1;
    }
    return 0;
}

extern "C" void fetch_output(std::complex<float> *outdata, DDCResources* res){
    int total_size = res->M * res->N;
    hipMemcpy(outdata, res->d_outdata, (total_size / res->NDEC) * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
}


extern "C" int calc_output_size(const DDCResources* res){
    return (res->M)*(res->N)/(res->NDEC);
}
