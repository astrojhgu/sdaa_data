#include "hip/hip_runtime.h"
#include "cuwf.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// CUDA 错误检查宏
#define CHECK_CUDA(call)                                                                 \
    do {                                                                                 \
        hipError_t err = (call);                                                        \
        if (err != hipSuccess) {                                                        \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,             \
                    hipGetErrorString(err));                                            \
            exit(EXIT_FAILURE);                                                          \
        }                                                                                \
    } while (0)

#define CHECK_CUFFT(call)                                                                \
    do {                                                                                 \
        hipfftResult err = (call);                                                        \
        if (err != HIPFFT_SUCCESS) {                                                      \
            fprintf(stderr, "CUFFT error at %s:%d: %d\n", __FILE__, __LINE__, err);      \
            exit(EXIT_FAILURE);                                                          \
        }                                                                                \
    } while (0)

struct Resource
{
    int nch;             // FFT点数为 2*nch
    int nbatch;          // 一次处理的batch数
    int nint;
    size_t total_length; // host缓冲区长度 = 2*nch*nbatch
    size_t filled;       // 已填充的数据点数（单位：int16_t个数）

    int16_t *host_buffer;  // host缓存
    int16_t *tmp_overflow; // 溢出缓存
    size_t tmp_len;        // 溢出缓存当前长度

    int16_t *d_raw_input;   // GPU上缓存的原始int16数据
    hipfftComplex *d_input;  // GPU输入 (complex<float>)
    hipfftComplex *d_output; // GPU输出
    float *d_spectrum;      // GPU上存储最终谱
    float *h_spectrum;      // host端谱输出缓冲区

    hipfftHandle fft_plan;
};

// CUDA kernel: int16_t → hipfftComplex (real part)，imag = 0
__global__ void convert_int16_to_complex(const int16_t* input, hipfftComplex* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx].x = static_cast<float>(input[idx]);
        output[idx].y = 0.0f;
    }
}

// CUDA kernel: 对每个频点，在每组 nint 条谱上做归约，输出 nch * (nbatch/nint) 个值
__global__ void compute_power_spectrum_grouped(
    const hipfftComplex* freq_data,
    float* spectrum,
    int nch,
    int nbatch,
    int nint)
{
    int freq_bin = blockIdx.x * blockDim.x + threadIdx.x;
    int group = blockIdx.y;  // 第 group 组
    if (freq_bin >= nch) return;

    float sum = 0.0f;
    int base_idx = group * nint;

    for (int j = 0; j < nint; ++j) {
        int idx = (base_idx + j) * (2 * nch) + freq_bin;
        hipfftComplex val = freq_data[idx];
        sum += val.x * val.x + val.y * val.y;
    }

    int out_idx = group * nch + freq_bin;
    spectrum[out_idx] = sum;
}


Resource* init_resource(int nch, int n_pt_per_payload, int nbatch, int nint) {
    Resource* res = (Resource*)malloc(sizeof(Resource));
    res->nch = nch;
    res->nint= nint;
    res->nbatch = nbatch;
    res->total_length = 2 * nch * nbatch;
    res->filled = 0;
    res->tmp_len = 0;

    res->host_buffer = (int16_t*)malloc(sizeof(int16_t) * res->total_length);
    res->tmp_overflow = (int16_t*)malloc(sizeof(int16_t) * n_pt_per_payload);
    res->h_spectrum = (float*)malloc(sizeof(float) * nch);

    CHECK_CUDA(hipMalloc(&res->d_raw_input, sizeof(int16_t) * res->total_length));
    CHECK_CUDA(hipMalloc(&res->d_input, sizeof(hipfftComplex) * res->total_length));
    CHECK_CUDA(hipMalloc(&res->d_output, sizeof(hipfftComplex) * res->total_length));
    CHECK_CUDA(hipMalloc(&res->d_spectrum, sizeof(float) * nch*nbatch/nint));

    CHECK_CUFFT(hipfftPlan1d(&res->fft_plan, 2 * nch, HIPFFT_C2C, nbatch));

    return res;
}

void destroy_resource(Resource* res) {
    free(res->host_buffer);
    free(res->tmp_overflow);
    free(res->h_spectrum);
    CHECK_CUDA(hipFree(res->d_raw_input));
    CHECK_CUDA(hipFree(res->d_input));
    CHECK_CUDA(hipFree(res->d_output));
    CHECK_CUDA(hipFree(res->d_spectrum));
    CHECK_CUFFT(hipfftDestroy(res->fft_plan));
    free(res);
}

bool waterfall(Resource* res, const int16_t* time_domain_input, size_t npt, float* output_spectrum) {
    if (res->filled + npt <= res->total_length) {
        memcpy(res->host_buffer + res->filled, time_domain_input, sizeof(int16_t) * npt);
        res->filled += npt;
        return false;
    }

    size_t first_part = res->total_length - res->filled;
    memcpy(res->host_buffer + res->filled, time_domain_input, sizeof(int16_t) * first_part);
    size_t remaining = npt - first_part;
    memcpy(res->tmp_overflow, time_domain_input + first_part, sizeof(int16_t) * remaining);
    res->tmp_len = remaining;
    res->filled = 0;

    int total_pts = res->total_length;
    int threads = 256;
    int blocks = (total_pts + threads - 1) / threads;

    CHECK_CUDA(hipMemcpy(res->d_raw_input, res->host_buffer,
                          sizeof(int16_t) * total_pts, hipMemcpyHostToDevice));

    convert_int16_to_complex<<<blocks, threads>>>(res->d_raw_input, res->d_input, total_pts);
    CHECK_CUDA(hipGetLastError());

    CHECK_CUFFT(hipfftExecC2C(res->fft_plan, res->d_input, res->d_output, HIPFFT_FORWARD));

    
    dim3 grid((res->nch + threads - 1) / threads, res->nbatch / res->nint);
    compute_power_spectrum_grouped<<<grid, threads>>>(res->d_output, res->d_spectrum, res->nch, res->nbatch, res->nint);
    CHECK_CUDA(hipDeviceSynchronize());

    
    // 主机端输出缓冲区大小也需要更新为 nch * (nbatch / nint)
    hipMemcpyAsync(output_spectrum, res->d_spectrum,
                    sizeof(float) * res->nch * (res->nbatch / res->nint),
                    hipMemcpyDeviceToHost);
    // 溢出数据回填到缓冲区
    memcpy(res->host_buffer, res->tmp_overflow, sizeof(int16_t) * res->tmp_len);
    res->filled = res->tmp_len;
    res->tmp_len = 0;

    return true;
}
